
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <string>
#include <unordered_map>
#include <fstream>
#include <algorithm>
#include <chrono>
#include <thread>

// Constants for scoring
const int MATCH_SCORE = 1;
const int GAP_COST = -1;
const int MISMATCH_COST = -1;
const int N_MAX=150;
const int TILE_SIZE = 150;
// Constants for Alignment tracebacking
// const int NONE = 0;
// const int DIAGONAL = 1;
// const int UP = 2;
// const int LEFT = 3;
__constant__ char* ConstQuery;

// Enum for choosing alignment algorithm
enum AlignmentAlgorithm { NEEDLEMAN_WUNSCH, SMITH_WATERMAN };
struct AlignmentResult {
    int score;
    std::string aligned_seq1;
    std::string aligned_seq2;
};

// Function to perform Needleman-Wunsch alignment
int needleman_wunsch(const std::string& seq1, const std::string& seq2) {
    int m = seq1.size();
    int n = seq2.size();
    std::vector<std::vector<int>> score_matrix(m + 1, std::vector<int>(n + 1, 0));

    // Initialize score matrix with gap penalties
    for (int i = 0; i <= m; ++i) score_matrix[i][0] = i * GAP_COST;
    for (int j = 0; j <= n; ++j) score_matrix[0][j] = j * GAP_COST;

    // Fill score matrix
    for (int i = 1; i <= m; ++i) {
        for (int j = 1; j <= n; ++j) {
            int match = score_matrix[i - 1][j - 1] + (seq1[i - 1] == seq2[j - 1] ? MATCH_SCORE : MISMATCH_COST);
            int delete_ = score_matrix[i - 1][j] + GAP_COST;
            int insert = score_matrix[i][j - 1] + GAP_COST;
        }
    }
    return score_matrix[m][n];  // Return global alignment score
}

// Kernel Function
__global__ void smith_waterman_kernel_batch(
    const char* seq1,
    const char* seq2,
    int m,
    int num_refs,
    int* score_matrices,
    int* max_scores,
    int* max_positions,
    char* aligned_seq1_array,
    char* aligned_seq2_array,
    int* alignment_lengths
)
{
    int n = N_MAX;

    int ref_string_idx = blockIdx.x;
    if(ref_string_idx >= num_refs) 
        return;

    __shared__ char s_ref_seq[N_MAX];

    int tid = threadIdx.x;

    const char* ref_seq = seq2 + ref_string_idx * N_MAX;
    int* score_matrix = score_matrices + ref_string_idx * (m+1) * (N_MAX + 1);
    int* max_score = max_scores + ref_string_idx;
    int* max_pos = max_positions + ref_string_idx * 2;

    // Pointers for alignment results
    char* aligned_seq1 = aligned_seq1_array + ref_string_idx * (m + N_MAX);  // Max possible alignment length
    char* aligned_seq2 = aligned_seq2_array + ref_string_idx * (m + N_MAX);
    int* alignment_length = alignment_lengths + ref_string_idx;

    // Load the reference sequence into shared memory
    for (int i = tid; i < N_MAX; i += blockDim.x) {
        s_ref_seq[i] = ref_seq[i];
    }
    __syncthreads();  // Ensure all threads have loaded the reference sequence

    for (int k = 2; k <= m + N_MAX; ++k) {
        int start_i = max(1, k - N_MAX);
        int end_i = min(m, k - 1);
        int num_elements = end_i - start_i + 1;

        for (int idx = tid; idx < num_elements; idx += blockDim.x) {
            int i = start_i + idx;
            int j = k - i;

            // Compute match/mismatch score
            int match = seq1[i - 1] == ref_seq[j - 1] ? MATCH_SCORE : MISMATCH_COST;

            // Calculate scores from neighboring cells
            int diag_score = score_matrix[(i - 1) * (N_MAX + 1) + (j - 1)] + match;
            int up_score = score_matrix[(i - 1) * (N_MAX + 1) + j] + GAP_COST;
            int left_score = score_matrix[i * (N_MAX + 1) + (j - 1)] + GAP_COST;

            int cell_score = max(0, max(diag_score, max(up_score, left_score)));

            score_matrix[i * (N_MAX + 1) + j] = cell_score;

             // Atomically update max score and position
            int old_max = atomicMax(max_score, cell_score);
            if (cell_score > old_max) {
                    atomicExch(&max_pos[0], i);
                    atomicExch(&max_pos[1], j);
                }
            }
            __syncthreads();
        }

        if(tid == 0)
        {
            int i = max_pos[0];
            int j = max_pos[1];
            int pos = 0;

            // Perform traceback until score is zero
            while (i > 0 && j > 0 && score_matrix[i * (n + 1) + j] > 0) {
                int current_score = score_matrix[i * (n + 1) + j];
                int diag_score = score_matrix[(i - 1) * (n + 1) + (j - 1)];
                int up_score = score_matrix[(i - 1) * (n + 1) + j];
                int left_score = score_matrix[i * (n + 1) + (j - 1)];

                if (current_score == diag_score + (seq1[i - 1] == s_ref_seq[j - 1] ? MATCH_SCORE : MISMATCH_COST)) {
                    aligned_seq1[pos] = seq1[i - 1];
                    aligned_seq2[pos] = s_ref_seq[j - 1];
                    --i;
                    --j;
                } else if (current_score == up_score + GAP_COST) {
                    aligned_seq1[pos] = seq1[i - 1];
                    aligned_seq2[pos] = '-';
                    --i;
                } else if (current_score == left_score + GAP_COST) {
                    aligned_seq1[pos] = '-';
                    aligned_seq2[pos] = s_ref_seq[j - 1];
                    --j;
                } else {
                    printf("Should not reach here in Smith-Waterman. Something is wrong");
                    break;
                }
                ++pos;
            }
            alignment_length[0] = pos;
            }
}




// Smith-Waterman alignment
std::vector<AlignmentResult> smith_waterman_batch(const std::string& h_seq1, const std::vector<std::string>& h_seq2_list) {
    int m = h_seq1.size();
    int num_refs = h_seq2_list.size();

    char* d_seq1;
    hipMalloc(&d_seq1, m * sizeof(char));
    hipMemcpy(d_seq1, h_seq1.data(), m * sizeof(char), hipMemcpyHostToDevice);

    char* d_seq2;
    int total_ref_len = N_MAX * num_refs;

    hipMalloc(&d_seq2, total_ref_len * sizeof(char));

    for(int i=0;i < num_refs;i++)
    {
        hipMemcpy(d_seq2+i*N_MAX,h_seq2_list[i].data(),N_MAX*sizeof(char),hipMemcpyHostToDevice); 
    }

    int* d_score_matrices;
    int score_matrix_size = (m + 1) * (N_MAX + 1);
    hipMalloc(&d_score_matrices, num_refs * score_matrix_size * sizeof(int));

    int* d_max_scores;
    int* d_max_positions;
    hipMalloc(&d_max_scores, num_refs * sizeof(int));
    hipMalloc(&d_max_positions, num_refs * 2 * sizeof(int));


    // Initialize score matrices and max scores/positions
    hipMemset(d_score_matrices, 0, num_refs * score_matrix_size * sizeof(int));
    hipMemset(d_max_scores, 0, num_refs * sizeof(int));
    hipMemset(d_max_positions, 0, num_refs * 2 * sizeof(int));

    //Device memory for alignment lengths
    char* d_aligned_seq1_arr;
    char* d_aligned_seq2_arr;
    int* d_alignment_lengths;
    int max_alignment_length = m + N_MAX;

    hipMalloc(&d_aligned_seq1_arr, num_refs * max_alignment_length * sizeof(char));
    hipMalloc(&d_aligned_seq2_arr, num_refs * max_alignment_length * sizeof(char));
    hipMalloc(&d_alignment_lengths, num_refs * sizeof(int));

    hipMemset(d_alignment_lengths, 0, num_refs * sizeof(int));

    int threadsPerBlock = N_MAX;
    int blocksPerGrid = num_refs;

    smith_waterman_kernel_batch<<<blocksPerGrid, threadsPerBlock>>>(
        d_seq1,
        d_seq2,
        m,
        num_refs,
        d_score_matrices,
        d_max_scores,
        d_max_positions,
        d_aligned_seq1_arr,
        d_aligned_seq2_arr,
        d_alignment_lengths
    );
    hipDeviceSynchronize();

    std::vector<int> h_max_scores(num_refs);
    std::vector<int> h_alignment_lengths(num_refs);
    std::vector<AlignmentResult> results(num_refs);

    hipMemcpy(h_max_scores.data(), d_max_scores, num_refs * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_alignment_lengths.data(), d_alignment_lengths, num_refs * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < num_refs; ++i) {
        int alignment_length = h_alignment_lengths[i];
        char* h_aligned_seq1 = (char*)malloc(alignment_length * sizeof(char));
        char* h_aligned_seq2 = (char*)malloc(alignment_length * sizeof(char));
    
        hipMemcpy(h_aligned_seq1, d_aligned_seq1_arr + i * max_alignment_length, alignment_length * sizeof(char), hipMemcpyDeviceToHost);
        hipMemcpy(h_aligned_seq2, d_aligned_seq2_arr + i * max_alignment_length, alignment_length * sizeof(char), hipMemcpyDeviceToHost);
    
        // Build the aligned sequences (reverse since they were constructed backwards)
        std::string aligned_seq1(h_aligned_seq1, alignment_length);
        std::string aligned_seq2(h_aligned_seq2, alignment_length);
        std::reverse(aligned_seq1.begin(), aligned_seq1.end());
        std::reverse(aligned_seq2.begin(), aligned_seq2.end());
    
        // Store the result
        results[i] = {h_max_scores[i], aligned_seq1, aligned_seq2};
    
        free(h_aligned_seq1);
        free(h_aligned_seq2);
    }

    hipFree(d_seq1);
    hipFree(d_seq2);
    hipFree(d_score_matrices);
    hipFree(d_max_scores);
    hipFree(d_max_positions);
    hipFree(d_alignment_lengths);

    return results;
}


// Hierarchical Index and Cache setup
std::unordered_map<char, std::vector<std::string>> sequence_index;
std::unordered_map<std::string, AlignmentResult> cache;

// Build a simple index based on the first character of sequences
void build_index(const std::vector<std::string>& sequences) {
    for (const auto& seq : sequences) {
        sequence_index[seq[0]].push_back(seq);
    }
}

// Search in index and use cache for faster access
AlignmentResult search_and_align(const std::string& query, AlignmentAlgorithm algorithm) {
    // Check cache
    if (cache.find(query) != cache.end()) {
        return cache[query];
    }

    char key = query[0];

    AlignmentResult final_result = {0,"",""};

    if (sequence_index.find(key) != sequence_index.end()) {

        std::vector<std::string>& ref_strings_matched = sequence_index[key];


        std::vector<AlignmentResult> results = smith_waterman_batch(query, ref_strings_matched);

        int max_score = -1;

        for (const auto& result : results) {
            if (result.score > max_score) {
                max_score = result.score;
                final_result = result;
            }
        }

    }

    // Store in cache
    cache[query] = final_result;

    return final_result;
    // return {max_score,final_aligned_seq1,final_aligned_seq2};
}

std::string alignment_markup(const std::string& seq1, const std::string& seq2) {
    std::string markup;
    for (size_t i = 0; i < seq1.length(); ++i) {
        if (seq1[i] == seq2[i]) {
            if (seq1[i] != '-') {  // Exclude gaps
                markup += "|";
            } else {
                markup += " ";
            }
        } else {
            markup += " ";
        }
    }
    return markup;
}

double calculate_percent_identity(const std::string& seq1, const std::string& seq2) {
    int matches = 0;
    int aligned_length = 0;
    for (size_t i = 0; i < seq1.length(); ++i) {
        if (seq1[i] != '-' && seq2[i] != '-') {
            aligned_length++;
            if (seq1[i] == seq2[i]) {
                matches++;
            }
        }
    }
    return (aligned_length > 0) ? (static_cast<double>(matches) / aligned_length) * 100.0 : 0.0;
}



// Function to simulate adaptive batching (for simplicity, it processes a batch size of 5)
void process_batches(const std::vector<std::string>& queries, AlignmentAlgorithm algorithm) {
    const int batch_size = 5;
    for (size_t i = 0; i < queries.size(); i += batch_size) {
        std::cout << "Processing batch " << (i / batch_size + 1) << "...\n";
        for (size_t j = i; j < std::min(i + batch_size, queries.size()); ++j) {
            AlignmentResult final_result = search_and_align(queries[j], algorithm);
            std::cout << "Query Sequence: " << queries[j] << "\n";
            std::cout << "Smith-Waterman alignment score: " << final_result.score << "\n";
            std::cout << "Aligned Query Sequence:      " << final_result.aligned_seq1 << "\n";
            std::cout << "                             " << alignment_markup(final_result.aligned_seq1, final_result.aligned_seq2) << "\n";
            std::cout << "Aligned Reference Sequence:  " << final_result.aligned_seq2 << "\n";
            double percent_identity = calculate_percent_identity(final_result.aligned_seq1, final_result.aligned_seq2);
            std::cout << "Percent Identity: " << percent_identity << "%\n";
            std::cout << "Alignment Length: " << final_result.aligned_seq1.length() << "\n\n";
        }
        std::this_thread::sleep_for(std::chrono::milliseconds(500));  // Simulate processing delay
    }
}

// Function to load sequences from a file
std::vector<std::string> load_sequences_from_file(const std::string& filename) {
    std::vector<std::string> sequences;
    std::ifstream file(filename);
    std::string line;
    while (std::getline(file, line)) {
        if (!line.empty()) {
            sequences.push_back(line);
        }
    }
    file.close();
    return sequences;
}

// Main function
int main() {
    // Load reference sequences from file
    std::vector<std::string> reference_sequences = load_sequences_from_file("proj_ref.txt");
    // Load query sequences from file
    std::vector<std::string> queries = load_sequences_from_file("proj_query.txt");

    // Build the hierarchical index
    build_index(reference_sequences);

    // Choose alignment algorithm (Needleman-Wunsch or Smith-Waterman)
    AlignmentAlgorithm algorithm = SMITH_WATERMAN;

    // Start timing
    auto start_time = std::chrono::high_resolution_clock::now();

    // Process the queries in adaptive batches
    process_batches(queries, algorithm);

    // Stop timing
    auto end_time = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> duration = end_time - start_time;

    // Calculate and print throughput
    double throughput = queries.size() / duration.count();
    std::cout << "Total Execution Time: " << duration.count() << " seconds\n";
    std::cout << "Throughput: " << throughput << " queries/second\n";
    std::cout << "Queries BLASTED: " << queries.size() << "\n";

    return 0;
}


