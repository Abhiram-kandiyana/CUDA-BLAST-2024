#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <string>
#include <unordered_map>
#include <fstream>
#include <algorithm>
#include <chrono>
#include <thread>

// Constants for scoring
const int MATCH_SCORE = 1;
const int GAP_COST = -1;
const int MISMATCH_COST = -1;
// Constants for Alignment tracebacking
// const int NONE = 0;
// const int DIAGONAL = 1;
// const int UP = 2;
// const int LEFT = 3;

// Enum for choosing alignment algorithm
enum AlignmentAlgorithm { NEEDLEMAN_WUNSCH, SMITH_WATERMAN };
struct AlignmentResult {
    int score;
    std::string aligned_seq1;
    std::string aligned_seq2;
};

// Function to perform Needleman-Wunsch alignment
int needleman_wunsch(const std::string& seq1, const std::string& seq2) {
    int m = seq1.size();
    int n = seq2.size();
    std::vector<std::vector<int>> score_matrix(m + 1, std::vector<int>(n + 1, 0));

    // Initialize score matrix with gap penalties
    for (int i = 0; i <= m; ++i) score_matrix[i][0] = i * GAP_COST;
    for (int j = 0; j <= n; ++j) score_matrix[0][j] = j * GAP_COST;

    // Fill score matrix
    for (int i = 1; i <= m; ++i) {
        for (int j = 1; j <= n; ++j) {
            int match = score_matrix[i - 1][j - 1] + (seq1[i - 1] == seq2[j - 1] ? MATCH_SCORE : MISMATCH_COST);
            int delete_ = score_matrix[i - 1][j] + GAP_COST;
            int insert = score_matrix[i][j - 1] + GAP_COST;
        }
    }
    return score_matrix[m][n];  // Return global alignment score
}

// Kernel Function
__global__ void compute_diagonal(int* score_matrix, const char* seq1, const char* seq2, int m, int n, int k, int* max_score, int* max_pos) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    int start_i = max(1, k - n);
    int end_i = min(m, k - 1);
    int num_elements = end_i - start_i + 1;

    if (idx >= num_elements) return;

    int i = start_i + idx;
    int j = k - i;

    // Compute match/mismatch score
    int match = seq1[i - 1] == seq2[j - 1] ? MATCH_SCORE : MISMATCH_COST;

    // Calculate scores from neighboring cells
    int diag_score = score_matrix[(i - 1) * (n + 1) + (j - 1)] + match;
    int up_score = score_matrix[(i - 1) * (n + 1) + j] + GAP_COST;
    int left_score = score_matrix[i * (n + 1) + (j - 1)] + GAP_COST;

    int cell_score = max(0, max(diag_score, max(up_score, left_score)));

    score_matrix[i * (n + 1) + j] = cell_score;

    // Atomically update max score and position
    int old_max = atomicMax(max_score, cell_score);
    if (cell_score > old_max) {
        atomicExch(&max_pos[0], i);
        atomicExch(&max_pos[1], j);
    }
}

// Updated Kernel Function
__global__ void smith_waterman_kernel(const char* seq1, const char* seq2, int m, int n,
    int* score_matrix, int* max_score, int* max_pos,
    char* aligned_seq1, char* aligned_seq2, int* alignment_length) {

    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Compute the score matrix in parallel
    for (int k = 2; k <= m + n; ++k) {
    int start_i = max(1, k - n);
    int end_i = min(m, k - 1);
    int num_elements = end_i - start_i + 1;

    if (idx < num_elements) {
    int i = start_i + idx;
    int j = k - i;

    // Compute match/mismatch score
    int match = seq1[i - 1] == seq2[j - 1] ? MATCH_SCORE : MISMATCH_COST;

    // Calculate scores from neighboring cells
    int diag_score = score_matrix[(i - 1) * (n + 1) + (j - 1)] + match;
    int up_score = score_matrix[(i - 1) * (n + 1) + j] + GAP_COST;
    int left_score = score_matrix[i * (n + 1) + (j - 1)] + GAP_COST;

    int cell_score = max(0, max(diag_score, max(up_score, left_score)));

    score_matrix[i * (n + 1) + j] = cell_score;

    // Atomically update max score and position
    int old_max = atomicMax(max_score, cell_score);
    if (cell_score > old_max) {
        atomicExch(&max_pos[0], i);
        atomicExch(&max_pos[1], j);
        }
    }
    __syncthreads();  // Ensure all threads have updated the score_matrix before next k
    }

    // Traceback (only one thread performs this)
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        int i = max_pos[0];
        int j = max_pos[1];
        int pos = 0;

        // Perform traceback until score is zero
        while (i > 0 && j > 0 && score_matrix[i * (n + 1) + j] > 0) {
            int current_score = score_matrix[i * (n + 1) + j];
            int diag_score = score_matrix[(i - 1) * (n + 1) + (j - 1)];
            int up_score = score_matrix[(i - 1) * (n + 1) + j];
            int left_score = score_matrix[i * (n + 1) + (j - 1)];

            if (current_score == diag_score + (seq1[i - 1] == seq2[j - 1] ? MATCH_SCORE : MISMATCH_COST)) {
                aligned_seq1[pos] = seq1[i - 1];
                aligned_seq2[pos] = seq2[j - 1];
                --i;
                --j;
            } 
            else if (current_score == up_score + GAP_COST) {
                aligned_seq1[pos] = seq1[i - 1];
                aligned_seq2[pos] = '-';
                --i;
            } 
            else if (current_score == left_score + GAP_COST) {
                aligned_seq1[pos] = '-';
                aligned_seq2[pos] = seq2[j - 1];
                --j;
            } else {
               printf("Should not reach here in Smith-Waterman. Something is wrong");
                break;
            }
            ++pos;
        }
        *alignment_length = pos;
    }
}



// Smith-Waterman alignment
AlignmentResult smith_waterman(const std::string& h_seq1, const std::string& h_seq2) {
    int m = h_seq1.size();
    int n = h_seq2.size();

    char* d_seq1;
    char* d_seq2;
    int* d_score_matrix;
    int* d_max_score;
    int* d_max_pos;
    char* d_aligned_seq1;
    char* d_aligned_seq2;
    int* d_alignment_length;
    int max_alignment_length = m + n;

    // Allocate device memory
    hipMalloc(&d_seq1, m * sizeof(char));
    hipMalloc(&d_seq2, n * sizeof(char));
    hipMalloc(&d_score_matrix, (m + 1) * (n + 1) * sizeof(int));
    hipMalloc(&d_max_score, sizeof(int));
    hipMalloc(&d_max_pos, 2 * sizeof(int));
    hipMalloc(&d_aligned_seq1, max_alignment_length * sizeof(char));
    hipMalloc(&d_aligned_seq2, max_alignment_length * sizeof(char));
    hipMalloc(&d_alignment_length, sizeof(int));
    hipMemset(d_alignment_length, 0, sizeof(int));

    // Copy sequences to device
    hipMemcpy(d_seq1, h_seq1.data(), m * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_seq2, h_seq2.data(), n * sizeof(char), hipMemcpyHostToDevice);

    // Initialize score matrix and max score/position
    hipMemset(d_score_matrix, 0, (m + 1) * (n + 1) * sizeof(int));
    hipMemset(d_max_score, 0, sizeof(int));
    hipMemset(d_max_pos, 0, 2 * sizeof(int));

    // Compute score matrix
    int threadsPerBlock = 256;
    int blocksPerGrid = (max(m, n) + threadsPerBlock - 1) / threadsPerBlock;
    smith_waterman_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_seq1, d_seq2, m, n,
        d_score_matrix, d_max_score, d_max_pos, d_aligned_seq1, d_aligned_seq2, d_alignment_length);

    hipDeviceSynchronize();

    // Copy alignment length back to host
    int h_alignment_length;
    int h_max_score;
    hipMemcpy(&h_alignment_length, d_alignment_length, sizeof(int), hipMemcpyDeviceToHost);

    // Allocate host memory for aligned sequences
    char* h_aligned_seq1 = (char*)malloc(h_alignment_length * sizeof(char));
    char* h_aligned_seq2 = (char*)malloc(h_alignment_length * sizeof(char));

    // Copy aligned sequences back to host
    hipMemcpy(h_aligned_seq1, d_aligned_seq1, h_alignment_length * sizeof(char), hipMemcpyDeviceToHost);
    hipMemcpy(h_aligned_seq2, d_aligned_seq2, h_alignment_length * sizeof(char), hipMemcpyDeviceToHost);
    hipMemcpy(&h_max_score, d_max_score,  sizeof(int), hipMemcpyDeviceToHost);

    // Build the aligned sequences (reverse since they were constructed backwards)
    std::string aligned_seq1(h_aligned_seq1, h_alignment_length);
    std::string aligned_seq2(h_aligned_seq2, h_alignment_length);
    std::reverse(aligned_seq1.begin(), aligned_seq1.end());
    std::reverse(aligned_seq2.begin(), aligned_seq2.end());

    // Free host memory for aligned sequences
    free(h_aligned_seq1);
    free(h_aligned_seq2);
    // Free device memory for aligned sequences and alignment length
    hipFree(d_aligned_seq1);
    hipFree(d_aligned_seq2);
    hipFree(d_alignment_length);

    return {h_max_score, aligned_seq1, aligned_seq2};
    // return {h_max_score, " ", " "};
}


// Hierarchical Index and Cache setup
std::unordered_map<char, std::vector<std::string>> sequence_index;
std::unordered_map<std::string, AlignmentResult> cache;

// Build a simple index based on the first character of sequences
void build_index(const std::vector<std::string>& sequences) {
    for (const auto& seq : sequences) {
        sequence_index[seq[0]].push_back(seq);
    }
}

// Search in index and use cache for faster access
AlignmentResult search_and_align(const std::string& query, AlignmentAlgorithm algorithm) {
    // Check cache
    if (cache.find(query) != cache.end()) {
        return cache[query];
    }

    // Search in index based on the first character
    char key = query[0];
    int max_score = 0;
    std::string final_aligned_seq1 ="";
    std::string final_aligned_seq2 ="";
    AlignmentResult result;
    AlignmentResult final_result;

    if (sequence_index.find(key) != sequence_index.end()) {
        for (const auto& seq : sequence_index[key]) {
            int score;
            if (algorithm == NEEDLEMAN_WUNSCH) {
                score = needleman_wunsch(query, seq);
            } else {  // SMITH_WATERMAN
                result = smith_waterman(query, seq);
            }
            if (result.score > max_score) {
                max_score = result.score;
                final_result = result;

            }
            // max_score = std::max(max_score, score);
        }
    }

    // Store in cache
    cache[query] = final_result;

    return final_result;
    // return {max_score,final_aligned_seq1,final_aligned_seq2};
}

std::string alignment_markup(const std::string& seq1, const std::string& seq2) {
    std::string markup;
    for (size_t i = 0; i < seq1.length(); ++i) {
        if (seq1[i] == seq2[i]) {
            if (seq1[i] != '-') {  // Exclude gaps
                markup += "|";
            } else {
                markup += " ";
            }
        } else {
            markup += " ";
        }
    }
    return markup;
}

double calculate_percent_identity(const std::string& seq1, const std::string& seq2) {
    int matches = 0;
    int aligned_length = 0;
    for (size_t i = 0; i < seq1.length(); ++i) {
        if (seq1[i] != '-' && seq2[i] != '-') {
            aligned_length++;
            if (seq1[i] == seq2[i]) {
                matches++;
            }
        }
    }
    return (aligned_length > 0) ? (static_cast<double>(matches) / aligned_length) * 100.0 : 0.0;
}



// Function to simulate adaptive batching (for simplicity, it processes a batch size of 5)
void process_batches(const std::vector<std::string>& queries, AlignmentAlgorithm algorithm) {
    const int batch_size = 5;
    for (size_t i = 0; i < queries.size(); i += batch_size) {
        std::cout << "Processing batch " << (i / batch_size + 1) << "...\n";
        for (size_t j = i; j < std::min(i + batch_size, queries.size()); ++j) {
            AlignmentResult final_result = search_and_align(queries[j], algorithm);
            std::cout << "Query Sequence: " << queries[j] << "\n";
            std::cout << "Smith-Waterman alignment score: " << final_result.score << "\n";
            std::cout << "Aligned Query Sequence:      " << final_result.aligned_seq1 << "\n";
            std::cout << "                             " << alignment_markup(final_result.aligned_seq1, final_result.aligned_seq2) << "\n";
            std::cout << "Aligned Reference Sequence:  " << final_result.aligned_seq2 << "\n";
            double percent_identity = calculate_percent_identity(final_result.aligned_seq1, final_result.aligned_seq2);
            std::cout << "Percent Identity: " << percent_identity << "%\n";
            std::cout << "Alignment Length: " << final_result.aligned_seq1.length() << "\n\n";
        }
        std::this_thread::sleep_for(std::chrono::milliseconds(500));  // Simulate processing delay
    }
}

// Function to load sequences from a file
std::vector<std::string> load_sequences_from_file(const std::string& filename) {
    std::vector<std::string> sequences;
    std::ifstream file(filename);
    std::string line;
    while (std::getline(file, line)) {
        if (!line.empty()) {
            sequences.push_back(line);
        }
    }
    file.close();
    return sequences;
}

// Main function
int main() {
    // Load reference sequences from file
    std::vector<std::string> reference_sequences = load_sequences_from_file("proj_ref.txt");
    // Load query sequences from file
    std::vector<std::string> queries = load_sequences_from_file("proj_query.txt");

    // Build the hierarchical index
    build_index(reference_sequences);

    // Choose alignment algorithm (Needleman-Wunsch or Smith-Waterman)
    AlignmentAlgorithm algorithm = SMITH_WATERMAN;

    // Start timing
    auto start_time = std::chrono::high_resolution_clock::now();

    // Process the queries in adaptive batches
    process_batches(queries, algorithm);

    // Stop timing
    auto end_time = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> duration = end_time - start_time;

    // Calculate and print throughput
    double throughput = queries.size() / duration.count();
    std::cout << "Total Execution Time: " << duration.count() << " seconds\n";
    std::cout << "Throughput: " << throughput << " queries/second\n";
    std::cout << "Queries BLASTED: " << queries.size() << "\n";

    return 0;
}


