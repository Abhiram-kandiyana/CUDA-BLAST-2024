
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <string>
#include <unordered_map>
#include <fstream>
#include <algorithm>
#include <chrono>
#include <thread>

// Constants for scoring
const int MATCH_SCORE = 1;
const int GAP_COST = -1;
const int MISMATCH_COST = -1;
// Constants for Alignment tracebacking
// const int NONE = 0;
// const int DIAGONAL = 1;
// const int UP = 2;
// const int LEFT = 3;

// Enum for choosing alignment algorithm
enum AlignmentAlgorithm { NEEDLEMAN_WUNSCH, SMITH_WATERMAN };
struct AlignmentResult {
    int score;
    std::string aligned_seq1;
    std::string aligned_seq2;
};

// Function to perform Needleman-Wunsch alignment
int needleman_wunsch(const std::string& seq1, const std::string& seq2) {
    int m = seq1.size();
    int n = seq2.size();
    std::vector<std::vector<int>> score_matrix(m + 1, std::vector<int>(n + 1, 0));

    // Initialize score matrix with gap penalties
    for (int i = 0; i <= m; ++i) score_matrix[i][0] = i * GAP_COST;
    for (int j = 0; j <= n; ++j) score_matrix[0][j] = j * GAP_COST;

    // Fill score matrix
    for (int i = 1; i <= m; ++i) {
        for (int j = 1; j <= n; ++j) {
            int match = score_matrix[i - 1][j - 1] + (seq1[i - 1] == seq2[j - 1] ? MATCH_SCORE : MISMATCH_COST);
            int delete_ = score_matrix[i - 1][j] + GAP_COST;
            int insert = score_matrix[i][j - 1] + GAP_COST;
        }
    }
    return score_matrix[m][n];  // Return global alignment score
}

// Kernel Function
__global__ void compute_diagonal(int* score_matrix, const char* seq1, const char* seq2, int m, int n, int k, int* max_score, int* max_pos) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    int start_i = max(1, k - n);
    int end_i = min(m, k - 1);
    int num_elements = end_i - start_i + 1;

    if (idx >= num_elements) return;

    int i = start_i + idx;
    int j = k - i;

    // Compute match/mismatch score
    int match = seq1[i - 1] == seq2[j - 1] ? MATCH_SCORE : MISMATCH_COST;

    // Calculate scores from neighboring cells
    int diag_score = score_matrix[(i - 1) * (n + 1) + (j - 1)] + match;
    int up_score = score_matrix[(i - 1) * (n + 1) + j] + GAP_COST;
    int left_score = score_matrix[i * (n + 1) + (j - 1)] + GAP_COST;

    int cell_score = max(0, max(diag_score, max(up_score, left_score)));

    score_matrix[i * (n + 1) + j] = cell_score;

    // Atomically update max score and position
    int old_max = atomicMax(max_score, cell_score);
    if (cell_score > old_max) {
        atomicExch(&max_pos[0], i);
        atomicExch(&max_pos[1], j);
    }
}

// Smith-Waterman alignment
AlignmentResult smith_waterman(const std::string& h_seq1, const std::string& h_seq2) {
    int m = h_seq1.size();
    int n = h_seq2.size();

    char* d_seq1;
    char* d_seq2;
    int* d_score_matrix;
    int* d_max_score;
    int* d_max_pos;

    // Allocate device memory
    hipMalloc(&d_seq1, m * sizeof(char));
    hipMalloc(&d_seq2, n * sizeof(char));
    hipMalloc(&d_score_matrix, (m + 1) * (n + 1) * sizeof(int));
    hipMalloc(&d_max_score, sizeof(int));
    hipMalloc(&d_max_pos, 2 * sizeof(int));

    // Copy sequences to device
    hipMemcpy(d_seq1, h_seq1.data(), m * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_seq2, h_seq2.data(), n * sizeof(char), hipMemcpyHostToDevice);

    // Initialize score matrix and max score/position
    hipMemset(d_score_matrix, 0, (m + 1) * (n + 1) * sizeof(int));
    hipMemset(d_max_score, 0, sizeof(int));
    hipMemset(d_max_pos, 0, 2 * sizeof(int));

    // Compute score matrix
    int threadsPerBlock = 256;
    for (int k = 2; k <= m + n; ++k) {
        int start_i = std::max(1, k - n);
        int end_i = std::min(m, k - 1);
        int num_elements = end_i - start_i + 1;

        int blocksPerGrid = (num_elements + threadsPerBlock - 1) / threadsPerBlock;
        compute_diagonal<<<blocksPerGrid, threadsPerBlock>>>(d_score_matrix, d_seq1, d_seq2, m, n, k, d_max_score, d_max_pos);
        hipDeviceSynchronize();
    }
    // Copy max score and position back to host
    int h_max_score;
    int h_max_pos[2];
    hipMemcpy(&h_max_score, d_max_score, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_max_pos, d_max_pos, 2 * sizeof(int), hipMemcpyDeviceToHost);

    int* h_score_matrix = (int*)malloc((m + 1) * (n + 1) * sizeof(int));
    hipMemcpy(h_score_matrix, d_score_matrix, (m + 1) * (n + 1) * sizeof(int), hipMemcpyDeviceToHost);


    std::string aligned_seq1 = "", aligned_seq2 = "";
    int i = h_max_pos[0];
    int j = h_max_pos[1];
    while (i > 0 && j > 0 && h_score_matrix[i * (n + 1) + j] > 0) {
        if (h_score_matrix[i * (n + 1) + j] == h_score_matrix[(i - 1) * (n + 1) + (j - 1)] + (h_seq1[i - 1] == h_seq2[j - 1] ? MATCH_SCORE : MISMATCH_COST)) {
            aligned_seq1 = h_seq1[i - 1] + aligned_seq1;
            aligned_seq2 = h_seq2[j - 1] + aligned_seq2;
            --i;
            --j;
        } else if (h_score_matrix[i * (n + 1) + j] == h_score_matrix[(i - 1) * (n + 1) + j] + GAP_COST) {
            aligned_seq1 = h_seq1[i - 1] + aligned_seq1;
            aligned_seq2 = "-" + aligned_seq2;
            --i;
        } else {
            aligned_seq1 = "-" + aligned_seq1;
            aligned_seq2 = h_seq2[j - 1] + aligned_seq2;
            --j;
        }
    }

    return {h_max_score, aligned_seq1, aligned_seq2};
    // return {h_max_score, " ", " "};
}


// Hierarchical Index and Cache setup
std::unordered_map<char, std::vector<std::string>> sequence_index;
std::unordered_map<std::string, AlignmentResult> cache;

// Build a simple index based on the first character of sequences
void build_index(const std::vector<std::string>& sequences) {
    for (const auto& seq : sequences) {
        sequence_index[seq[0]].push_back(seq);
    }
}

// Search in index and use cache for faster access
AlignmentResult search_and_align(const std::string& query, AlignmentAlgorithm algorithm) {
    // Check cache
    if (cache.find(query) != cache.end()) {
        return cache[query];
    }

    // Search in index based on the first character
    char key = query[0];
    int max_score = 0;
    std::string final_aligned_seq1 ="";
    std::string final_aligned_seq2 ="";
    AlignmentResult result;
    AlignmentResult final_result;

    if (sequence_index.find(key) != sequence_index.end()) {
        for (const auto& seq : sequence_index[key]) {
            int score;
            if (algorithm == NEEDLEMAN_WUNSCH) {
                score = needleman_wunsch(query, seq);
            } else {  // SMITH_WATERMAN
                result = smith_waterman(query, seq);
            }
            if (result.score > max_score) {
                max_score = result.score;
                final_result = result;

            }
            // max_score = std::max(max_score, score);
        }
    }

    // Store in cache
    cache[query] = final_result;

    return final_result;
    // return {max_score,final_aligned_seq1,final_aligned_seq2};
}

std::string alignment_markup(const std::string& seq1, const std::string& seq2) {
    std::string markup;
    for (size_t i = 0; i < seq1.length(); ++i) {
        if (seq1[i] == seq2[i]) {
            if (seq1[i] != '-') {  // Exclude gaps
                markup += "|";
            } else {
                markup += " ";
            }
        } else {
            markup += " ";
        }
    }
    return markup;
}

double calculate_percent_identity(const std::string& seq1, const std::string& seq2) {
    int matches = 0;
    int aligned_length = 0;
    for (size_t i = 0; i < seq1.length(); ++i) {
        if (seq1[i] != '-' && seq2[i] != '-') {
            aligned_length++;
            if (seq1[i] == seq2[i]) {
                matches++;
            }
        }
    }
    return (aligned_length > 0) ? (static_cast<double>(matches) / aligned_length) * 100.0 : 0.0;
}



// Function to simulate adaptive batching (for simplicity, it processes a batch size of 5)
void process_batches(const std::vector<std::string>& queries, AlignmentAlgorithm algorithm) {
    const int batch_size = 5;
    for (size_t i = 0; i < queries.size(); i += batch_size) {
        std::cout << "Processing batch " << (i / batch_size + 1) << "...\n";
        for (size_t j = i; j < std::min(i + batch_size, queries.size()); ++j) {
            AlignmentResult final_result = search_and_align(queries[j], algorithm);
            std::cout << "Query Sequence: " << queries[j] << "\n";
            std::cout << "Smith-Waterman alignment score: " << final_result.score << "\n";
            std::cout << "Aligned Query Sequence:      " << final_result.aligned_seq1 << "\n";
            std::cout << "                             " << alignment_markup(final_result.aligned_seq1, final_result.aligned_seq2) << "\n";
            std::cout << "Aligned Reference Sequence:  " << final_result.aligned_seq2 << "\n";
            double percent_identity = calculate_percent_identity(final_result.aligned_seq1, final_result.aligned_seq2);
            std::cout << "Percent Identity: " << percent_identity << "%\n";
            std::cout << "Alignment Length: " << final_result.aligned_seq1.length() << "\n\n";
        }
        std::this_thread::sleep_for(std::chrono::milliseconds(500));  // Simulate processing delay
    }
}

// Function to load sequences from a file
std::vector<std::string> load_sequences_from_file(const std::string& filename) {
    std::vector<std::string> sequences;
    std::ifstream file(filename);
    std::string line;
    while (std::getline(file, line)) {
        if (!line.empty()) {
            sequences.push_back(line);
        }
    }
    file.close();
    return sequences;
}

// Main function
int main() {
    // Load reference sequences from file
    std::vector<std::string> reference_sequences = load_sequences_from_file("proj_ref.txt");
    // Load query sequences from file
    std::vector<std::string> queries = load_sequences_from_file("proj_query.txt");

    // Build the hierarchical index
    build_index(reference_sequences);

    // Choose alignment algorithm (Needleman-Wunsch or Smith-Waterman)
    AlignmentAlgorithm algorithm = SMITH_WATERMAN;

    // Start timing
    auto start_time = std::chrono::high_resolution_clock::now();

    // Process the queries in adaptive batches
    process_batches(queries, algorithm);

    // Stop timing
    auto end_time = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> duration = end_time - start_time;

    // Calculate and print throughput
    double throughput = queries.size() / duration.count();
    std::cout << "Total Execution Time: " << duration.count() << " seconds\n";
    std::cout << "Throughput: " << throughput << " queries/second\n";

    return 0;
}


